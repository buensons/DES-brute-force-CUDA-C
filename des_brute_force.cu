#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <strings.h>

#include "hip/hip_runtime.h"
#include ""

#include "des_cpu_functions.cuh"
#include "des_gpu_functions.cuh"

#define ERR(source) (perror(source), fprintf(stderr,"%s:%d\n",__FILE__,__LINE__), exit(EXIT_FAILURE))

int main(int argc, char ** argv) {

    // could-have: take input message and split into 64-bit blocks
    uint64 data = 0x0123456789ABCDEF;

    if(argc != 2) {
        printf("Usage: %s <key_size>\n", argv[0]);
        return EXIT_FAILURE;
    }

    int key_size = atoi(argv[1]);
    if(key_size > 64) {
        printf("Key size reduced to 64 bits.");
        key_size = 64;
    }
    uint64 key = generate_key(key_size);
    uint64 encrypted_message = encrypt_message(data, key);
    clock_t start, end;
    float time_elapsed;

    // --------- CPU -------------

    printf("CPU : Brute forcing DES...\n");
    
    start = clock();

    for(uint64 i = 0; i <= ~(0ULL); i++) {
        uint64 msg = encrypt_message(data, i);
        //printBits(i);
        if(msg == encrypted_message) {
            end = clock();
            time_elapsed = ((float) (end - start)) / CLOCKS_PER_SEC;
            printf("CPU : Key found!\n");
            printf("CPU : Found key: %llX\n", i);
            printf("CPU : Time elapsed - %f\n", time_elapsed);
            break;
        }
    }


    // --------- GPU ------------
    
    //hipSetDevice(cutGetMaxGflopsDeviceId());

    int * has_key = NULL;
    int temp = 0;
    uint64 * cracked_key = NULL;
    uint64 found_key;

    hipError_t error;

    if((error = hipMalloc(&has_key, sizeof(int))) != hipSuccess) {
        ERR(hipGetErrorString(error));
    }

    if((error = hipMalloc(&cracked_key, sizeof(uint64))) != hipSuccess) {
        ERR(hipGetErrorString(error));
    }

    if((error = hipMemcpy(has_key, &temp, sizeof(int), hipMemcpyHostToDevice)) != hipSuccess) {
        ERR(hipGetErrorString(error));
    }

    printf("\nGPU : Brute forcing DES...\n");
    start = clock();

    brute_force<<<4096, 1024>>>(data, encrypted_message, cracked_key, has_key);

    end = clock();
    time_elapsed = ((float) (end - start)) / CLOCKS_PER_SEC;

    if((error = hipDeviceSynchronize()) != hipSuccess) ERR(hipGetErrorString(error));
    
    if((error = hipMemcpy(&found_key, cracked_key, sizeof(uint64), hipMemcpyDeviceToHost)) != hipSuccess) {
        ERR(hipGetErrorString(error));
    }

    printf("GPU : Key found!\n");
    printf("GPU : Time elapsed - %f\n", time_elapsed);
    printf("GPU : Cracked key: %llX\n", found_key);

    hipFree(has_key);
    hipFree(cracked_key);

    return EXIT_SUCCESS;
}
