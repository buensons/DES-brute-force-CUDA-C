#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <strings.h>

#include "hip/hip_runtime.h"
#include ""

#include "des_cpu_functions.cuh"
#include "des_gpu_functions.cuh"

#define DES_KEY_SIZE 64

#define ERR(source) (perror(source), fprintf(stderr,"%s:%d\n",__FILE__,__LINE__), exit(EXIT_FAILURE))

int main() {

    // change it later and add padding with 0's if mod 64 != 0
    uint64 data = 0x0123456789ABCDEF;
    uint64 key = generate_key();
    uint64 encrypted_message = encrypt_message(data, key);

    // --------- CPU -------------

    printf("CPU : Brute forcing DES...\n");

    uint64 i = 0;

    for(i = ~(i); i >= 0; i--) {
        uint64 msg = encrypt_message(data, i);
        //printBits(i);
        if(msg == encrypted_message) {
            printf("CPU : Key found!\n");
            printf("CPU : Original Key: %llX\n", key);
            printf("CPU : Found key: %llX\n", i);
            break;
        }
    }

    // --------- CUDA ------------
    
    //hipSetDevice(cutGetMaxGflopsDeviceId());

    bool * has_key = NULL;
    bool temp = false;
    uint64 * cracked_key = NULL;
    uint64 found_key;

    hipError_t error;

    if((error = hipMalloc(has_key, sizeof(bool))) != hipSuccess) {
        ERR(hipGetErrorString(error));
    }

    if((error = hipMalloc(cracked_key, sizeof(uint64))) != hipSuccess) {
        ERR(hipGetErrorString(error));
    }

    if((error = hipMemcpy(has_key, &temp, sizeof(bool), hipMemcpyHostToDevice)) != hipSuccess) {
        ERR(hipGetErrorString(error));
    }

    printf("GPU : Brute forcing DES...\n");
    brute_force<<<4096, 1024>>>(data, encrypted_message, cracked_key, has_key);

    if((error = hipDeviceSynchronize()) != hipSuccess) ERR(hipGetErrorString(error));
    
    if((error = hipMemcpy(&found_key, cracked_key, sizeof(uint64), hipMemcpyDeviceToHost)) != hipSuccess) {
        ERR(hipGetErrorString(error));
    }

    printf("GPU : Key found!\n");
    printf("GPU : Time elapsed - ");
    printf("GPU : Cracked key: %llX\n", found_key);

    hipFree(has_key);
    hipFree(cracked_key);

    return 0;
}
