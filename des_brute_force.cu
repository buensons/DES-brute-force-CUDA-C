#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <strings.h>

#include "hip/hip_runtime.h"
#include ""

#include "des_cpu_functions.cuh"
#include "des_gpu_functions.cuh"

#define ERR(source) (perror(source), fprintf(stderr,"%s:%d\n",__FILE__,__LINE__), exit(EXIT_FAILURE))

int main(int argc, char ** argv) {

    // could-have: take input message and split into 64-bit blocks
    uint64 data = 0x0123456789ABCDEF;

    if(argc != 2) {
        perror("Usage: %s <key_size>\n", argv[0]);
        return EXIT_FAILURE;
    }

    int key_size = atoi(argv[1]);
    uint64 key = generate_key(key_size);
    uint64 encrypted_message = encrypt_message(data, key);
    clock_t start, end;
    double time_elapsed;

    // --------- GPU ------------
    
    //hipSetDevice(cutGetMaxGflopsDeviceId());

    bool * has_key = NULL;
    bool temp = false;
    uint64 * cracked_key = NULL;
    uint64 found_key;

    hipError_t error;

    if((error = hipMalloc(&has_key, sizeof(bool))) != hipSuccess) {
        ERR(hipGetErrorString(error));
    }

    if((error = hipMalloc(&cracked_key, sizeof(uint64))) != hipSuccess) {
        ERR(hipGetErrorString(error));
    }

    if((error = hipMemcpy(has_key, &temp, sizeof(bool), hipMemcpyHostToDevice)) != hipSuccess) {
        ERR(hipGetErrorString(error));
    }

    printf("GPU : Brute forcing DES...\n");
    start = clock();

    brute_force<<<4096, 1024>>>(data, encrypted_message, cracked_key, has_key);

    end = clock();
    time_elapsed = ((double) (end - start)) / CLOCKS_PER_SEC;

    if((error = hipDeviceSynchronize()) != hipSuccess) ERR(hipGetErrorString(error));
    
    if((error = hipMemcpy(&found_key, cracked_key, sizeof(uint64), hipMemcpyDeviceToHost)) != hipSuccess) {
        ERR(hipGetErrorString(error));
    }

    printf("GPU : Key found!\n");
    printf("GPU : Time elapsed - %d\n", time_elapsed);
    printf("GPU : Cracked key: %llX\n", found_key);

    hipFree(has_key);
    hipFree(cracked_key);


    // --------- CPU -------------

    printf("CPU : Brute forcing DES...\n");
    
    start = clock();

    for(uint64 i = 0; i <= ~(0ULL); i++) {
        uint64 msg = encrypt_message(data, i);
        //printBits(i);
        if(msg == encrypted_message) {
            end = clock();
            time_elapsed = ((double) (end - start)) / CLOCKS_PER_SEC;
            printf("CPU : Key found!\n");
            printf("CPU : Found key: %llX\n", i);
            printf("CPU : Time elapsed - %d\n", time_elapsed);
            break;
        }
    }

    return EXIT_SUCCESS;
}
